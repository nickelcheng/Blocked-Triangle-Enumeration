
#include <hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>
#include<vector>
#include<algorithm>
#include<sys/time.h>

#define swap(a,b) {int tmp = a; a = b, b = tmp;}

#define cntTime(st,ed)\
((double)ed.tv_sec*1000000+ed.tv_usec-(st.tv_sec*1000000+st.tv_usec))/1000

#define timerInit(n)\
struct timeval st[n], ed[n];

#define timerStart(n)\
gettimeofday(&st[n], NULL);

#define timerEnd(tar, n)\
gettimeofday(&ed[n], NULL);\
fprintf(stderr, "%s: %.3lf ms\n", tar, cntTime(st[n],ed[n]));
//fprintf(stderr, " %.3lf", cntTime(st[n],ed[n]));


using namespace std;

typedef struct Node Node;
typedef struct Edge Edge;
typedef struct Triangle Triangle;

struct Node{
    vector< int > largerDegNei;
    int realDeg;
    int newOrder;
    Node(void){
        realDeg = 0;
        largerDegNei.clear();
    }
    void addNei(int v){
        largerDegNei.push_back(v);
    }
    int degree(void) const{
        return (int)largerDegNei.size();
    }
};

struct Edge{
    int u, v;
    Edge(int _u, int _v){
        u = _u, v = _v;
    }
};

struct Triangle{
    int a, b, c;
    Triangle(int _a, int _b, int _c){
        a = _a, b = _b, c = _c;
    }
    bool operator < (const Triangle &t) const{
        if(a != t.a) return a < t.a;
        if(b != t.b) return b < t.b;
        return c < t.c;
    }
    void sortNode(void){
        if(a > b) swap(a,b);
        if(a > c) swap(a,c);
        if(b > c) swap(b,c);
    }
};

void input(const char *inFile, vector< Node > &node, vector< Edge > &edge);
void reorderByDegree(vector< Node > &node, vector< Edge > &edge);
void updateGraph(vector< Node > &node, vector< Edge > &edge);
__global__ void countTriNum(int *offset, int *edgeV, int *triNum, int nodeNum);
__device__ int intersectList(int sz1, int sz2, int *l1, int *l2);

extern __shared__ int shared[]; // adj[maxDeg], threadTriNum[threadNum]

int main(int argc, char *argv[]){
    if(argc != 5){
        fprintf(stderr, "usage: forward <input_path> <node_num> <thread_per_block> <block_num>\n");
        return 0;
    }

    timerInit(2)
    timerStart(0)

    int nodeNum = atoi(argv[2]);
    vector< Node > node(nodeNum);
    vector< Edge > edge;

    timerStart(1)
    input(argv[1], node, edge);
    timerEnd("input", 1)

    timerStart(1)
    reorderByDegree(node, edge);
    updateGraph(node, edge);
    timerEnd("reordering", 1)

    int maxDeg = 0;
    for(int i = 0; i < nodeNum; i++){
        if(node[i].degree() > maxDeg)
            maxDeg = node[i].degree();
    }

    int edgeNum = (int)edge.size();
    int triNum = 0, *h_offset, *h_edgeV;
    int *d_triNum, *d_offset, *d_edgeV;
    
    h_offset = (int*)malloc(sizeof(int)*(nodeNum+1));
    h_edgeV = (int*)malloc(sizeof(int)*edgeNum);

    h_offset[0] = 0;
    for(int i = 0; i < nodeNum; i++){
        int deg = node[i].degree();
        h_offset[i+1] = h_offset[i] + deg;
        for(int j = 0; j < deg; j++){
            int idx = h_offset[i] + j;
            h_edgeV[idx] = node[i].largerDegNei[j];
        }
    }

    timerStart(1)
    hipMalloc((void**)&d_triNum, sizeof(int));
    hipMalloc((void**)&d_offset, sizeof(int)*(nodeNum+1));
    hipMalloc((void**)&d_edgeV, sizeof(int)*edgeNum);

    hipMemcpy(d_triNum, &triNum, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offset, h_offset, sizeof(int)*(nodeNum+1), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeV, h_edgeV, sizeof(int)*edgeNum, hipMemcpyHostToDevice);
    timerEnd("cuda copy", 1)

    timerStart(1)
    int threadNum = atoi(argv[3]);
    int blockNum = atoi(argv[4]);
    int smSize = (threadNum+maxDeg) * sizeof(int);
/*    printf("%d blocks, %d threads per block\n", nodeNum, threadNum);
    printf("smSize = %d bytes\n", smSize);*/
    countTriNum<<< blockNum, threadNum, smSize >>>(d_offset, d_edgeV, d_triNum, nodeNum);
    hipDeviceSynchronize();
    timerEnd("intersection", 1)

    hipMemcpy(&triNum, d_triNum, sizeof(int), hipMemcpyDeviceToHost);
    printf("total triangle: %d\n", triNum);

    hipFree(d_triNum);
    hipFree(d_offset);
    hipFree(d_edgeV);

    free(h_offset);
    free(h_edgeV);

    timerEnd("total", 0)

    return 0;
}

void input(const char *inFile, vector< Node > &node, vector< Edge > &edge){
    FILE *fp = fopen(inFile, "r");
    int u, v;
    while(fscanf(fp, "%d%d", &u, &v) != EOF){
        edge.push_back(Edge(u,v));
    }
    fclose(fp);
}

void reorderByDegree(vector< Node > &node, vector< Edge > &edge){
    int nodeNum = (int)node.size();
    int edgeNum = (int)edge.size();
    vector< vector< int > > degList(nodeNum);

    // count degree for each node
    for(int i = 0; i < edgeNum; i++){
        node[edge[i].u].realDeg++;
        node[edge[i].v].realDeg++;
    }
    // reorder by counting sort
    for(int i = 0; i < nodeNum; i++){
        degList[node[i].realDeg].push_back(i);
    }
    for(int i = 0, deg = 0; deg < nodeNum; deg++){
        for(int j = 0; j < (int)degList[deg].size(); j++){
            node[degList[deg][j]].newOrder = i++;
        }
    }
}

void updateGraph(vector< Node > &node, vector< Edge > &edge){
    int edgeNum = (int)edge.size();
    int nodeNum = (int)node.size();

    for(int i = 0; i < edgeNum; i++){
        edge[i].u = node[edge[i].u].newOrder;
        edge[i].v = node[edge[i].v].newOrder;
    }

    for(int i = 0; i < edgeNum; i++){
        int u = edge[i].u, v = edge[i].v;
        if(u < v) node[u].addNei(v);
        else node[v].addNei(u);
    }

    for(int i = 0; i < nodeNum; i++){
        sort(node[i].largerDegNei.begin(), node[i].largerDegNei.end());
    }
}

__global__ void countTriNum(int *offset, int *edgeV, int *triNum, int nodeNum){
    int nodePerBlock = (int)ceil((double)nodeNum/gridDim.x-0.001);
    for(int r = 0; r < nodePerBlock; r++){
        int nodeID = blockIdx.x*nodePerBlock + r;
        if(nodeID >= nodeNum) continue;
        int myOffset = offset[nodeID];
        int nextOffset = offset[nodeID+1];
        int deg = nextOffset - myOffset;
        int jobPerThread = (int)ceil((double)deg/blockDim.x-0.001);

        // move node u's adj list to shared memory
        for(int i = 0; i < jobPerThread; i++){
            int idx = threadIdx.x*jobPerThread + i;
            if(idx < deg){
                shared[idx] = edgeV[myOffset+idx]; // adj[idx]
            }
        }
        __syncthreads();

        // counting triangle number
        shared[deg+threadIdx.x] = 0;
        for(int i = 0; i < jobPerThread; i++){
            int idx = threadIdx.x*jobPerThread + i;
            if(idx < deg){
                int v = shared[idx]; // adj[idx]
                int vNeiLen = offset[v+1] - offset[v];
                shared[deg+threadIdx.x] += intersectList(deg, vNeiLen, shared, &edgeV[offset[v]]); // threadTriNum[threadIdx.x]
            }
        }
        __syncthreads();

        // sum triangle number
        if(threadIdx.x == 0){
            int tmp = 0;
            for(int i = 0; i < blockDim.x; i++){
                tmp += shared[deg+i]; // threadTriNum[i]
            }
            atomicAdd(triNum, tmp);
        }
    }
}

__device__ int intersectList(int sz1, int sz2, int *l1, int *l2){
    int i, j;
    int triNum = 0;
    for(i = sz1-1, j = sz2-1; i >= 0 && j >= 0;){
        if(l1[i] > l2[j]) i--;
        else if(l1[i] < l2[j]) j--;
        else{
            i--, j--;
            triNum++;
        }
    }
    return triNum;
}
